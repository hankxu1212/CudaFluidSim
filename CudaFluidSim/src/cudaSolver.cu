#include "hip/hip_runtime.h"
#include "Solver.hpp"
#include "math/Math.hpp"
#include "cudaSolver.cuh"

#define GLM_ENABLE_EXPERIMENTAL
#include "glm/gtx/norm.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <>

#include "Particle.hpp"

#define WINDOW_HEIGHT Window::Get()->m_Data.Height
#define WINDOW_WIDTH Window::Get()->m_Data.Width
#define m_Particles Solver::Get()->m_Particles

__constant__ glm::vec2 cuG;
__constant__ int cuH;
__constant__ int cuCELL_SIZE;
__constant__ float cuHSQ;
__constant__ float cuREST_DENS;
__constant__ float cuGAS_CONST;
__constant__ float cuMASS;
__constant__ float cuVISC;
__constant__ float cuDT;
__constant__ float cuPOLY6;
__constant__ float cuSPIKY_GRAD;
__constant__ float cuVISC_LAP;
__constant__ float cuEPS;
__constant__ float cuBOUND_DAMPING;
__constant__ unsigned int cuWINDOW_WIDTH;
__constant__ unsigned int cuWINDOW_HEIGHT;


__global__ void KernelComputeDensityPressure(Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i];
		float density = 0.f;
		for (int j = 0; j < N; ++j)
		{
			auto& pj = particles[j];
			glm::vec2 rij = pj.position - pi.position;
			float r2 = glm::length2(rij);

			if (r2 < cuHSQ)
			{
				// this computation is symmetric
				pi.density += cuMASS * cuPOLY6 * pow(cuHSQ - r2, 3.f);
			}
		}
		pi.pressure = cuGAS_CONST * (pi.density - cuREST_DENS);
	}
}

__global__ void KernelComputeForces(Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i];

		glm::vec2 fpress(0.f, 0.f);
		glm::vec2 fvisc(0.f, 0.f);
		for (int j = 0; j < N; ++j)
		{
			auto& pj = particles[j];

			if (&pi == &pj)
			{
				continue;
			}

			glm::vec2 rij = pj.position - pi.position;
			float r = glm::length(rij);

			if (r < H)
			{
				// compute pressure force contribution
				float mangitude = cuMASS * (pi.pressure + pj.pressure) / (2.0f * pj.density) * cuSPIKY_GRAD * pow(H - r, 3);
				fpress += glm::normalize(-rij) * mangitude;

				// compute viscosity force contribution
				fvisc += cuVISC * cuMASS * (pj.velocity - pi.velocity) / pj.density * cuVISC_LAP * (H - r);
			}
		}
		glm::vec2 fgrav = cuG * cuMASS / pi.density;
		pi.force = fpress + fvisc + fgrav;
	}
}

__global__ void KernelIntegrate(float dt, Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& p = particles[i];

		// forward Euler integration
		p.velocity += dt * p.force / p.density;
		p.position += dt * p.velocity;

		// enforce boundary conditions
		if (p.position[0] - cuEPS < 0.f)
		{
			p.velocity[0] *= cuBOUND_DAMPING;
			p.position[0] = cuEPS;
		}
		if (p.position[0] + cuEPS > cuWINDOW_WIDTH)
		{
			p.velocity[0] *= cuBOUND_DAMPING;
			p.position[0] = cuWINDOW_WIDTH - cuEPS;
		}
		if (p.position[1] - cuEPS < 0.f)
		{
			p.velocity[1] *= cuBOUND_DAMPING;
			p.position[1] = cuEPS;
		}
		if (p.position[1] + cuEPS > cuWINDOW_HEIGHT)
		{
			p.velocity[1] *= cuBOUND_DAMPING;
			p.position[1] = cuWINDOW_HEIGHT - cuEPS;
		}
	}
}

void CUDAInitSPH()
{
	for (int i = 0; i < NUM_PARTICLES; ++i)
	{
		float angle = Math::Random(0, 2.0f * 3.1415f);
		float r = Math::Random();

		float x = WINDOW_HEIGHT / 3 + WINDOW_HEIGHT / 3 * r * cos(angle) + WINDOW_HEIGHT / 5;
		float y = WINDOW_HEIGHT / 3 + WINDOW_HEIGHT / 3 * r * sin(angle);

		m_Particles[i] = Particle(x, y);
	}

	std::cout << "Initializing " << m_Particles.size() << " particles" << std::endl;

	hipMalloc(&d_Particles, m_Particles.size() * sizeof(Particle));
	hipMemcpy(d_Particles, m_Particles.data(), m_Particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(&cuG), &G, sizeof(glm::vec2));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuH), &H, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuCELL_SIZE), &CELL_SIZE, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuHSQ), &HSQ, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuREST_DENS), &REST_DENS, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuGAS_CONST), &GAS_CONST, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuMASS), &MASS, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuVISC), &VISC, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuDT), &DT, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuPOLY6), &POLY6, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuSPIKY_GRAD), &SPIKY_GRAD, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuVISC_LAP), &VISC_LAP, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuEPS), &EPS, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuBOUND_DAMPING), &BOUND_DAMPING, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuWINDOW_HEIGHT), &WINDOW_HEIGHT, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuWINDOW_WIDTH), &WINDOW_WIDTH, sizeof(unsigned int));
}

void DispatchComputeDensityPressure()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeDensityPressure <<<gridDim, blockDim>>> (d_Particles, m_Particles.size());
	// probably doesn't properly compile and stuff, but am focusing on writing the logic before figuring out how
	// the syntax works
}

void DispatchComputeForces()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeForces <<<gridDim, blockDim>>> (d_Particles, m_Particles.size());
}

void DispatchIntegrate(float dt)
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelIntegrate <<<gridDim, blockDim>>> (dt, d_Particles, m_Particles.size());
}