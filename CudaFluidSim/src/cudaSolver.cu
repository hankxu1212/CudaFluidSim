#include "hip/hip_runtime.h"
#include "cudaSolver.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <>

// host handles
d_Particle* d_Particles;

// device constants
//__constant__ float2 cuG;
//__constant__ uint32_t cuWINDOW_WIDTH;
//__constant__ uint32_t cuWINDOW_HEIGHT;

__global__ void KernelComputeDensityPressure(d_Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i].position_density_pressure;
		float density = 0.f;
		for (int j = 0; j < NUM_PARTICLES; ++j)
		{
			auto& pj = particles[j].position_density_pressure;
			float2 rij = make_float2(pj.x - pi.x, pj.y - pi.y);
			float r2 = rij.x*rij.x + rij.y*rij.y;

			if (r2 < HSQ)
			{
				// this computation is symmetric
				density += MASS * POLY6 * pow(HSQ - r2, 3.f);
			}
		}
		pi.z = density;
		pi.w = GAS_CONST * (density - REST_DENS);
	}
}

__global__ void KernelComputeForces(d_Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i];

		float2 fpress = make_float2(0.f, 0.f);
		float2 fvisc = make_float2(0.f, 0.f);
		for (int j = 0; j < N; ++j)
		{
			auto& pj = particles[j];

			if (&pi == &pj) { continue; }

			float2 rij = make_float2(pj.position_density_pressure.x - pi.position_density_pressure.x, pj.position_density_pressure.y - pi.position_density_pressure.y);
			float r = sqrt(rij.x * rij.x + rij.y * rij.y);

			if (r < H)
			{
				// compute pressure force contribution
				float magnitude = MASS * (pi.position_density_pressure.w + pj.position_density_pressure.w) / (2.0f * pj.position_density_pressure.z) * SPIKY_GRAD * pow(H - r, 3.f);
				fpress = make_float2(fpress.x - rij.x/r * magnitude, fpress.y - rij.y/r * magnitude);

				// compute viscosity force contribution
				fvisc = make_float2(
					fvisc.x + VISC * MASS * (pj.velocity_force.x - pi.velocity_force.x) / pj.position_density_pressure.z * VISC_LAP * (H - r),
					fvisc.y + VISC * MASS * (pj.velocity_force.y - pi.velocity_force.y) / pj.position_density_pressure.z * VISC_LAP * (H - r)
				);
			}
		}
		float2 fgrav = make_float2(cuGx * MASS / pi.position_density_pressure.z, cuGy * MASS / pi.position_density_pressure.z);
		pi.velocity_force.z = fpress.x + fvisc.x + fgrav.x;
		pi.velocity_force.w = fpress.y + fvisc.y + fgrav.y;
	}
}

__global__ void KernelIntegrate(float dt, d_Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& p = particles[i];

		// forward Euler integration
		p.velocity_force.x += dt * p.velocity_force.z / p.position_density_pressure.z;
		p.velocity_force.y += dt * p.velocity_force.w / p.position_density_pressure.z;
		p.position_density_pressure.x += dt * p.velocity_force.x;
		p.position_density_pressure.y += dt * p.velocity_force.y;

		// enforce boundary conditions
		if (p.position_density_pressure.x - EPS < 0.f)
		{
			p.velocity_force.x *= BOUND_DAMPING;
			p.position_density_pressure.x = EPS;
		}
		if (p.position_density_pressure.x + EPS > cuWINDOW_WIDTH)
		{
			p.velocity_force.x *= BOUND_DAMPING;
			p.position_density_pressure.x = cuWINDOW_WIDTH - EPS;
		}
		if (p.position_density_pressure.y - EPS < 0.f)
		{
			p.velocity_force.y *= BOUND_DAMPING;
			p.position_density_pressure.y = EPS;
		}
		if (p.position_density_pressure.y + EPS > cuWINDOW_HEIGHT)
		{
			p.velocity_force.y *= BOUND_DAMPING;
			p.position_density_pressure.y = cuWINDOW_HEIGHT - EPS;
		}
	}
}

void DeviceInitSPH(d_Particle* hostParticles)
{
	hipMalloc(&d_Particles, NUM_PARTICLES * sizeof(d_Particle));
	hipMemcpy(d_Particles, hostParticles, NUM_PARTICLES * sizeof(d_Particle), hipMemcpyHostToDevice);

	//hipMemcpyToSymbol(HIP_SYMBOL(&cuG), &G, sizeof(glm::vec2));
}

void DispatchComputeDensityPressure()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeDensityPressure <<<gridDim, blockDim>>> (d_Particles, NUM_PARTICLES);
	// probably doesn't properly compile and stuff, but am focusing on writing the logic before figuring out how
	// the syntax works
}

void DispatchComputeForces()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeForces <<<gridDim, blockDim>>> (d_Particles, NUM_PARTICLES);
}

void DispatchIntegrate(float dt)
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelIntegrate <<<gridDim, blockDim>>> (dt, d_Particles, NUM_PARTICLES);
}

void DeviceCleanup()
{
	if (d_Particles != nullptr)
		hipFree(d_Particles);
}

void DeviceSync(d_Particle* hostParticles, size_t count)
{
	hipMemcpy(hostParticles, d_Particles, count * sizeof(d_Particle), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}
