#include "hip/hip_runtime.h"
#include "cudaSolver.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <>

// host handles
Particle* d_Particles;

// device constants
__constant__ glm::vec2 cuG;
__constant__ unsigned int cuWINDOW_WIDTH;
__constant__ unsigned int cuWINDOW_HEIGHT;

__global__ void KernelComputeDensityPressure(Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i];
		float density = 0.f;
		for (int j = 0; j < NUM_PARTICLES; ++j)
		{
			auto& pj = particles[j];
			glm::vec2 rij = pj.position - pi.position;
			float r2 = glm::length2(rij);

			if (r2 < HSQ)
			{
				// this computation is symmetric
				pi.density += MASS * POLY6 * pow(HSQ - r2, 3.f);
			}
		}
		pi.pressure = GAS_CONST * (pi.density - REST_DENS);
	}
}

__global__ void KernelComputeForces(Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& pi = particles[i];

		glm::vec2 fpress(0.f, 0.f);
		glm::vec2 fvisc(0.f, 0.f);
		for (int j = 0; j < N; ++j)
		{
			auto& pj = particles[j];

			if (&pi == &pj)
			{
				continue;
			}

			glm::vec2 rij = pj.position - pi.position;
			float r = glm::length(rij);

			if (r < H)
			{
				// compute pressure force contribution
				float mangitude = MASS * (pi.pressure + pj.pressure) / (2.0f * pj.density) * SPIKY_GRAD * pow(H - r, 3);
				fpress += glm::normalize(-rij) * mangitude;

				// compute viscosity force contribution
				fvisc += VISC * MASS * (pj.velocity - pi.velocity) / pj.density * VISC_LAP * (H - r);
			}
		}
		glm::vec2 fgrav = cuG * MASS / pi.density;
		pi.force = fpress + fvisc + fgrav;
	}
}

__global__ void KernelIntegrate(float dt, Particle* particles, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		auto& p = particles[i];

		// forward Euler integration
		p.velocity += dt * p.force / p.density;
		p.position += dt * p.velocity;

		// enforce boundary conditions
		if (p.position[0] - EPS < 0.f)
		{
			p.velocity[0] *= BOUND_DAMPING;
			p.position[0] = EPS;
		}
		if (p.position[0] + EPS > cuWINDOW_WIDTH)
		{
			p.velocity[0] *= BOUND_DAMPING;
			p.position[0] = cuWINDOW_WIDTH - EPS;
		}
		if (p.position[1] - EPS < 0.f)
		{
			p.velocity[1] *= BOUND_DAMPING;
			p.position[1] = EPS;
		}
		if (p.position[1] + EPS > cuWINDOW_HEIGHT)
		{
			p.velocity[1] *= BOUND_DAMPING;
			p.position[1] = cuWINDOW_HEIGHT - EPS;
		}
	}
}

void DeviceInitSPH(Particle* hostParticles, uint32_t windowHeight, uint32_t windowWidth)
{
	hipMalloc(&d_Particles, NUM_PARTICLES * sizeof(Particle));
	hipMemcpy(d_Particles, hostParticles, NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(&cuG), &G, sizeof(glm::vec2));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuWINDOW_HEIGHT), &windowHeight, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(&cuWINDOW_WIDTH), &windowWidth, sizeof(unsigned int));
}

void DispatchComputeDensityPressure()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeDensityPressure <<<gridDim, blockDim>>> (d_Particles, NUM_PARTICLES);
	// probably doesn't properly compile and stuff, but am focusing on writing the logic before figuring out how
	// the syntax works
}

void DispatchComputeForces()
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelComputeForces <<<gridDim, blockDim>>> (d_Particles, NUM_PARTICLES);
}

void DispatchIntegrate(float dt)
{
	dim3 blockDim(256, 1);
	dim3 gridDim((NUM_PARTICLES + blockDim.x - 1) / blockDim.x);
	KernelIntegrate <<<gridDim, blockDim>>> (dt, d_Particles, NUM_PARTICLES);
}

void DeviceCleanup()
{
	if (d_Particles != nullptr)
		hipFree(d_Particles);
}

void DeviceSync(Particle* hostParticles, size_t count)
{
	hipMemcpy(hostParticles, d_Particles, count * sizeof(Particle), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}
